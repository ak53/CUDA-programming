
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <unistd.h>

#define LENGTH 10000000
using namespace std;

__global__ void vector_add(float *a, float *b, float *c){
	int index = threadIdx.x + blockDim.x * blockIdx.x; 
	if (index<LENGTH){
		c[index] = a[index] + b[index];
	}
}

void myCpu(){
	unsigned int microseconds = 10000000;
	usleep(microseconds);
}

int main(){

	float *a_vec, *b_vec, *c_vec;

	a_vec = (float*)malloc(LENGTH*sizeof(float));
	b_vec = (float*)malloc(LENGTH*sizeof(float));

	c_vec = (float*)malloc(LENGTH*sizeof(float)); //cpu device -> host

	for(int i=0 ; i< LENGTH; i++){
		a_vec[i] = i;
		b_vec[i] = i;
	}

	float *d_a, *d_b, *d_c;

	hipMalloc((void**)&d_a, LENGTH*sizeof(float));
	hipMalloc((void**)&d_b, LENGTH*sizeof(float));
	hipMalloc((void**)&d_c, LENGTH*sizeof(float)); // host -> device

	hipMemcpy(d_a, a_vec, LENGTH*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b_vec, LENGTH*sizeof(float), hipMemcpyHostToDevice);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	vector_add<<<(int)ceil((float)LENGTH/1024),1024>>>(d_a, d_b, d_c); //what happens if no of threads becomes decimal
	myCpu();
	// cudaDeviceSynchronize();
	hipEventRecord(stop);

	hipMemcpy(c_vec, d_c, LENGTH*sizeof(float), hipMemcpyDeviceToHost);
	
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	std::cout  << "Time taken : " << milliseconds << std::endl;
	std::cout<<"First 3 elements are "<<c_vec[0]<<"  "<<c_vec[1]<<"  "<<c_vec[2]<<'\n';
	free(a_vec);
	free(b_vec);
	free(c_vec);

	// for(int i=0; i<LENGTH ;i++){
	// 	cout << c_vec[i] << endl;
	// }
}
