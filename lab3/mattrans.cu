// Author: Amandeep Kaur (2018014)
// As part of Lab3 in CSE:560 GPU computing course


#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

#define size 10000
#define bs 16
#define mem_row 8

__global__ void trans(int* mat, int* transmat){
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	if (row<size && col<size){
		// for (int i=0;i<bs;i+=mem_row){
			transmat[(row) + size*(col)] = mat[(row)*size + col];
		// }
	}
}

int main(){
	hipEvent_t start_gpu, end_gpu;
	float msecs_gpu;
	hipEventCreate(&start_gpu);
	hipEventCreate(&end_gpu);
	hipEventRecord(start_gpu, 0);

	int *mat;
	hipError_t status = hipHostMalloc((void**)&mat, size*size*sizeof(int), hipHostMallocDefault);
	// mat = (int*)malloc(size*size*sizeof(int));
	if (status!=hipSuccess){
		cout<<"Error occured in pinned memory"<<endl;
		return 0;
	}
	for (int i=0;i<size;i++){
		for (int j=0;j<size;j++){
			mat[i*size+j]=j;
		}
	}
	int *dmat;
	hipMalloc((void**)&dmat, size*size*sizeof(int));
	hipMemcpy(dmat, mat,size*size*sizeof(int), hipMemcpyHostToDevice );
	// size_t pitch;
	// status = cudaMallocPitch((void**)&dmat, &pitch, size*sizeof(int),size);
	// if (status!=cudaSuccess){
	// 	cout<<"Error occured in PITCH initialise"<<endl;
	// 	return 0;
	// }
	// status = cudaMemcpy2D(dmat,pitch, mat,size*sizeof(int), pitch, size, cudaMemcpyHostToDevice);
	// if (status!=cudaSuccess){
	// 	cout<<"Error occured in PITCH copy"<<endl;
	// 	cout<<cudaGetErrorString(status)<<endl;
	// 	return 0;
	// }


	int *transmat;
	// transmat = (int*)malloc(size*size*sizeof(int));
	status = hipHostMalloc((void**)&transmat, size*size*sizeof(int), hipHostMallocDefault);
	if (status!=hipSuccess){
		cout<<"Error occured in pinned memory"<<endl;
		return 0;
	}

	int *dtransmat;
	hipMalloc((void**)&dtransmat, size*size*sizeof(int));


	dim3 grid,block;
	block.x = bs;
	block.y = bs;
	grid.x = ((int)size/bs)+1;
	grid.y = ((int)size/bs)+1;
	cout<<"Safely reached"<<endl;
	hipEvent_t start, end;
	float msecs;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start, 0);

	trans<<<grid,block>>>(dmat, dtransmat);

	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&msecs, start, end);
	hipEventDestroy(start);
	hipEventDestroy(end);
	cout<<"kernel done in "<<msecs<<" milliseconds.\n";

	hipMemcpy(transmat, dtransmat, size*size*sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(end_gpu, 0);
	hipEventSynchronize(end_gpu);
	hipEventElapsedTime(&msecs_gpu, start_gpu, end_gpu);
	hipEventDestroy(start_gpu);
	hipEventDestroy(end_gpu);
	cout<<"done in "<<msecs_gpu<<" milliseconds.\n";

	// for (int i=size-5;i<size;i++){
	// 	for (int j=0;j<size;j++){
	// 		cout<<transmat[i*size+j]<<"  ";
	// 	}
	// 	cout<<endl;
	// }

	hipFree(dmat);
	hipHostFree(mat);
	hipFree(dtransmat);
	hipHostFree(transmat);
	return 0;
}