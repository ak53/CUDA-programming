#include "hip/hip_runtime.h"
#include <float.h>
#include <stdio.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <thrust/scan.h>
#include "helper.h"

using std::ofstream;

__global__ void translateAndMapPixels(int* map, int num_pixel, float* points, int n_points, float minX, float minY, int cols)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < n_points){
		int x = (int)(points[2*idx]-minX+1.0);
		int y = (int)(points[2*idx+1]-minY+1.0);
		if (x<0 || y<0)
			printf("(%i,%i) Point outside grid\n",x,y);
		else if (y*cols+x < num_pixel){
			map[y*cols+x] = idx;
		}
	}
}

__global__ void voronoiDiagram(int* map, int rows, int cols, float* points, int n_points, int stepsize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = 0;
    if (x<cols && y<rows){
    	int dx[] = {-1,0,1};
    	int dy[] = {-1,0,1};
    	idx = map[y*cols+x];
    	for (int i=0; i<3; ++i){
    		for (int j=0; j<3; ++j){
    			if (i==0 && j==0)
    				continue;	
				int nx = x+dx[i]*stepsize, ny = y+dy[j]*stepsize;
				if (nx>=cols || nx<0 || ny>=rows || ny<0)
					continue;
				int nIdx = map[ny*cols+nx];
				if (nIdx == -1)
					continue;
				if (idx == -1)
					idx = nIdx;
				else{
					int xd2 = points[2*nIdx]-x, yd2 = points[2*nIdx+1]-y;
					int xd1 = points[2*idx]-x, yd1 = points[2*idx+1]-y;
					if (xd2*xd2+yd2*yd2 <= xd1*xd1+yd1*yd1)
						idx = nIdx;
				}
    		}
    	}
    }
	__syncthreads();
	if (x<cols && y<rows)
		map[y*cols+x] = idx;
}

__global__ void count_triangles(int* map, int rows, int cols, int* count){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x<cols-1 && y<rows-1){
    	int a = map[y*cols+x], b = map[y*cols+x+1], c = map[(y+1)*cols+x], d = map[(y+1)*cols+x+1], val=0;
    	if (a!=b && b!=c && a!=c && a!=d && b!=d && c!=d)
			val = 2;	
		else if ((a!=b && b!=c && a!=c)||(a!=d && d!=c && a!=c)||(b!=d && d!=c && b!=c))
			val = 1;
		count[y*cols+x] = val;
    }
}

__global__ void triangulate(int* map, int* count, int* total_cnt, int rows, int cols, int* triangles, int n_triangles){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x<cols-1 && y<rows-1){
    	int val = count[y*cols+x];
    	if (val > 0){
	    	int id = total_cnt[y*cols+x];
	    	int a=map[y*cols+x], b=map[y*cols+x+1], c=map[(y+1)*cols+x], d=map[(y+1)*cols+x+1];
	    	if (val == 1){
	    		if (a==b)	b=c;
				if (b==c)	c=d;
	    		triangles[3*id] = a;
	    		triangles[3*id+1] = b;
	    		triangles[3*id+2] = c;
	    	}
	    	else if (val == 2){
	    		triangles[3*id] = a;
	    		triangles[3*id+1] = b;
	    		triangles[3*id+2] = c;
	    		triangles[3*id+3] = b;
	    		triangles[3*id+4] = c;
	    		triangles[3*id+5] = d;
	    	}
    	}
    }
}

void computeDT_GPU_(float* points, int n_points, float* bounds, int* triangles, int &n_triangles)
{
	int threads = 32;
	float minX = bounds[0], minY = bounds[1], maxX = bounds[2], maxY = bounds[3];

	int rows = (int)(maxY-minY+2);
	int cols = (int)(maxX-minX+2);

	int *d_map, *tri_count, *total_cnt, *d_triangles;
	float *d_points;

	hipEvent_t start_1, stop_1;
	hipEventCreate(&start_1);
	hipEventCreate(&stop_1);
	hipMalloc((void**)&d_map, rows*cols*sizeof(int));
	hipMalloc((void**)&d_points, 2*n_points*sizeof(float));

	hipMemcpy(d_points, points, 2*n_points*sizeof(float), hipMemcpyHostToDevice);
	hipMemset(d_map, -1, rows*cols*sizeof(int));

	translateAndMapPixels<<<(n_points+threads-1)/threads, threads>>>(d_map, rows*cols, d_points, n_points, minX, minY, cols);
	hipDeviceSynchronize();
	

	dim3 blockDim(threads,threads);
	dim3 gridDim((cols+threads-1)/threads,(rows+threads-1)/threads);

	int stepsize = np2(min(rows,cols))/2;
	hipEventRecord(start_1);
	while (stepsize >= 1){
		voronoiDiagram<<<gridDim, blockDim>>>(d_map, rows, cols, d_points, n_points, stepsize);
		hipDeviceSynchronize();
		stepsize /= 2;
	}
	hipEventRecord(stop_1);

	int *map = (int*)malloc(sizeof(int)*rows*cols);
	hipMemcpy(map, d_map, sizeof(int)*rows*cols, hipMemcpyDeviceToHost);
	ofstream outdata; // outdata is like cin
	outdata.open("gpu.txt"); // opens the file
	if( !outdata ) { // file couldn't be opened
		std::cerr << "Error: file could not be opened" << endl;
		exit(1);
	}
	else{
		for (int i = 0; i < rows; ++i){
			for (int j = 0; j < cols; ++j)
				outdata << map[i*cols+j] <<",";
			outdata << "\n";
		}
		outdata.close();
	}

	hipMalloc((void**)&tri_count, rows*cols*sizeof(int));
	count_triangles<<<gridDim, blockDim>>>(d_map, rows, cols, tri_count);
	hipDeviceSynchronize();
	

	hipMalloc((void**)&total_cnt, rows*cols*sizeof(int));
	thrust::inclusive_scan(thrust::device, tri_count, tri_count + rows*cols, total_cnt);
	hipDeviceSynchronize();
	hipMemcpy(&n_triangles, &total_cnt[rows*cols-1], sizeof(int), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop_1);


	hipMalloc((void**)&d_triangles, 3*n_triangles*sizeof(int));
	triangles = (int*)malloc(3*n_triangles*sizeof(int));
	
	triangulate<<<gridDim, blockDim>>>(d_map, tri_count, total_cnt, rows, cols, d_triangles, n_triangles);
	hipDeviceSynchronize();
	hipMemcpy(triangles, d_triangles, 3*n_triangles*sizeof(int), hipMemcpyDeviceToHost);

	float ms_1 = 0;
	hipEventElapsedTime(&ms_1, start_1, stop_1);
	printf("Computation took %f milliseconds on GPU.\n",ms_1);

	hipFree(d_points);	
	hipFree(d_map);
	hipFree(tri_count);
	hipFree(total_cnt);
	hipFree(d_triangles);
}


