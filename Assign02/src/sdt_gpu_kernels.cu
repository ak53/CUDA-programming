
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <float.h>

#define tile 4096

__global__ void sdt_gpu(unsigned char * bitmap,int sz_edge, int* edge_pixels, float *sdt, int width, int height)
{
  __shared__ int s[tile];
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int bdx = blockDim.x;
  int global_idx = bx * bdx + tx;

  int iter_on_edge_pixels = (sz_edge + tile - 1)/tile;
  float min_dist, dist2;
  float _x, _y;
  float sign;
  float dx, dy;
  int x, y, k;

  min_dist = FLT_MAX;

  for (int p=0;p<iter_on_edge_pixels;p++){
    int base = p*tile;
    //to handle iterations on shared memory
    //if sz_edge is smaller; iterate till sz_edge
    //else tile
    int end = tile;
    if (sz_edge<base+end){
      end=sz_edge%tile;
    }

    // if sz_edge is smaller
    // if (tx<sz_edge) s[tx] = edge_pixels[base+tx];
    for (int i=0;i<tile;i+=bdx){

      if (base+i+tx<sz_edge) s[i+tx] = edge_pixels[base+i+tx];
    }
    __syncthreads();

    if (global_idx<height*width){
      x = global_idx%width;
      y = global_idx/width;
      for (k=0;k<end;k++){
        int q = s[k]; //bank conflicts but same data is fetched
        _x = q%width;
        _y = q/width;
        dx = _x-x;
        dy =_y-y;
        dist2 = dx*dx + dy*dy;
        if (dist2<min_dist) min_dist=dist2; 
      }
    }
    __syncthreads();
  }
  sign = (bitmap[global_idx] >= 127)? 1.0f : -1.0f;
  sdt[global_idx] = sign * sqrtf(min_dist);

}


extern "C" void run_sampleKernel(unsigned char * bitmap, float *sdt, int width, int height)
{
  unsigned char * d_bitmap;
  float * d_sdt;
  int *d_edge;

  hipMalloc((void**)&d_bitmap, height*width*sizeof(unsigned char));
  hipMalloc((void**)&d_sdt, height*width*sizeof(float));

  hipMemcpy(d_bitmap,bitmap, height*width*sizeof(unsigned char),hipMemcpyHostToDevice);

  // INITIALIZING OUTPUT IMAGE SPACE IN DEVICE MEMORY

  int sz = width*height;
  int sz_edge = 0;
  for(int i = 0; i<sz; i++) if(bitmap[i] == 255) sz_edge++;
  int *edge_pixels = new int[sz_edge];
  for(int i = 0, j = 0; i<sz; i++) if(bitmap[i] == 255) edge_pixels[j++] = i;
  std::cout<< "\t"<<sz_edge << " edge pixels in the image of size " << width << " x " << height << "\n"<<std::flush;
  hipMalloc((void**)&d_edge,sz_edge*sizeof(int));
  hipMemcpy(d_edge, edge_pixels, sz_edge*sizeof(int), hipMemcpyHostToDevice);
  std::cout<<"Calling kernel"<<std::endl;


  hipEvent_t start_gpu, end_gpu;
  float msecs_gpu;
  hipEventCreate(&start_gpu);
  hipEventCreate(&end_gpu);
  hipEventRecord(start_gpu, 0);


  sdt_gpu<<<(sz+1024-1)/1024, 1024>>>(d_bitmap,sz_edge, d_edge, d_sdt,width,height);

  hipDeviceSynchronize();


  hipEventRecord(end_gpu, 0);
  hipEventSynchronize(end_gpu);
  hipEventElapsedTime(&msecs_gpu, start_gpu, end_gpu);
  hipEventDestroy(start_gpu);
  hipEventDestroy(end_gpu);
  std::cout<<"\tComputation took "<<msecs_gpu<<" milliseconds.\n";
  hipMemcpy(sdt, d_sdt, sz*sizeof(float),hipMemcpyDeviceToHost);
  hipFree(d_sdt);
  hipFree(d_edge);
  hipFree(d_bitmap);
  free(edge_pixels);
}