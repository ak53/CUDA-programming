// Author: Amandeep Kaur (2018014)
// As part of assignment 1 in CSE:560 GPU computing course
// Code adapted from given code file ahe_cpu.cpp provided by Prof. Ojaswa Sharma


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <time.h>

#define TILE_SIZE_X 1024
#define TILE_SIZE_Y 1024

__constant__ unsigned char const_mappings[65536];

__global__ void findEqualizationMappings(unsigned char* img_in, int width, int height, unsigned char *mappings, int *pdf, int *cdf)
{
	int ntiles_x = (width / TILE_SIZE_X);

	int ty = threadIdx.y;
	int tx = threadIdx.x;
	int by = blockIdx.y;
	int bx = blockIdx.x;

	int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    
    if (row<height && col<width){
		int tile_i = (col)/TILE_SIZE_X; //0-indexed
		int tile_j = (row)/TILE_SIZE_Y;
		int offset = 256*(tile_i + tile_j * ntiles_x);
		atomicAdd(&pdf[offset + img_in[col+row*width]],1);
		__syncthreads();

		if ((row+1)%TILE_SIZE_Y==0 && (col + 1)%TILE_SIZE_X==0){ //one thread from each block
			int cdf_min = TILE_SIZE_X*TILE_SIZE_Y+1; // minimum non-zero value 
			cdf[offset]=pdf[offset];
			for(int i=1; i< 256; i++)
				cdf[offset+i] = cdf[offset+i-1] + pdf[offset+i];
			for(int i=0; i<256; i++)
				if(cdf[offset+i] != 0) {cdf_min = cdf[offset+i]; break;}
		
			for (int i=0;i<256;i++){
				mappings[i + offset] = (unsigned char)round(255.0 * float(cdf[offset+i] - cdf_min)/float(TILE_SIZE_X*TILE_SIZE_Y - cdf_min));
				}
			}
	}
}

__global__ void performAdaptiveEqualization(unsigned char* img_in, unsigned char* img_out, int width, int height){

	int ntiles_x = (width / TILE_SIZE_X);
	int ntiles_y = (height / TILE_SIZE_Y);

	int ty = threadIdx.y;
	int tx = threadIdx.x;
	int by = blockIdx.y;
	int bx = blockIdx.x;

	int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    if (row<height && col<width){

		// FINDING TILE CENTERS FOR INTERPOLATION
		int tile_i0, tile_j0, tile_i1, tile_j1;
		tile_i0 = (col - TILE_SIZE_X/2) / TILE_SIZE_X;
		if(tile_i0 < 0) tile_i0 = 0;
		tile_j0 = (row - TILE_SIZE_Y/2) / TILE_SIZE_Y;
		if(tile_j0 < 0) tile_j0 = 0;
		tile_i1 = (col + TILE_SIZE_X/2) / TILE_SIZE_X;
		if(tile_i1 >= ntiles_x) tile_i1 = ntiles_x - 1;
		tile_j1 = (row + TILE_SIZE_Y/2) / TILE_SIZE_Y;
		if(tile_j1 >= ntiles_y) tile_j1 = ntiles_y - 1;

		// OFFSETS IN INTERMEDIATE ARRAYS CORRESPONDING TO TILE CENTERS
		int offset00 = 256*(tile_i0 + tile_j0*ntiles_x);
		int offset01 = 256*(tile_i0 + tile_j1*ntiles_x);
		int offset10 = 256*(tile_i1 + tile_j0*ntiles_x);
		int offset11 = 256*(tile_i1 + tile_j1*ntiles_x);

	    unsigned char v00, v01, v10, v11;
		v00 = const_mappings[img_in[col+row*width] + offset00];
		v01 = const_mappings[img_in[col+row*width] + offset01];
		v10 = const_mappings[img_in[col+row*width] + offset10];
		v11 = const_mappings[img_in[col+row*width] + offset11];

		float x_frac = float(col - tile_i0*TILE_SIZE_X - TILE_SIZE_X/2)/float(TILE_SIZE_X);
		float y_frac = float(row - tile_j0*TILE_SIZE_Y - TILE_SIZE_Y/2)/float(TILE_SIZE_Y);
		
		//PERFORMING BILINEAR INTERPOLATION
	  	float v0 = v00*(1 - x_frac) + v10*x_frac;
		float v1 = v01*(1 - x_frac) + v11*x_frac;
	    float v= v0*(1 - y_frac) + v1*y_frac;

		if (v < 0) v = 0;
		if (v > 255) v = 255;

	    img_out[col+row*width] = (unsigned char)(v);
	}
}

extern "C" void run_sampleKernel(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
  int ntiles_x = (width / TILE_SIZE_X);
  int ntiles_y = (height / TILE_SIZE_Y);
  int ntiles = (ntiles_x * ntiles_y);

// INITIALIZING REQUIREMENTS
  int *dpdf;
  int *dcdf;
  hipMalloc((void**)&dpdf, 256*ntiles*sizeof(int));
  hipMalloc((void**)&dcdf, 256*ntiles*sizeof(int));
  hipMemset(dpdf, 0, 256*ntiles*sizeof(int));
  unsigned char *dmappings;
  hipMalloc((void**)&dmappings, 256*ntiles*sizeof(unsigned char));

// WRITING INPUT IMAGE TO DEVICE MEMORY
  unsigned char * dimg_in;
  hipMalloc((void**)&dimg_in, height*width*sizeof(unsigned char));
  hipMemcpy(dimg_in,img_in, height*width*sizeof(unsigned char),hipMemcpyHostToDevice);
  
// INITIALIZING OUTPUT IMAGE SPACE IN DEVICE MEMORY
  unsigned char * dimg_out;
  hipMalloc((void**)&dimg_out, height*width*sizeof(unsigned char));

// SETTING UP LAUNCH CONFIGURATION
  dim3 grid,block;
  block.x = 32;
  block.y = 32;
  int req = (height*width)/(32*32);
  grid.x = pow(req,0.5);
  grid.y = pow(req,0.5);

// TIMER
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start);

  //STEP 1
  findEqualizationMappings<<<grid,block>>>(dimg_in, width, height, dmappings, dpdf, dcdf);
  
 // cudaDeviceSynchronize();
 
 // COPYING MAPPINGS TO CONST_MAPPINGS (TO USE CONSTANT MEMORY)
 int *mappings;
 mappings = (int *)malloc(256*ntiles*sizeof(unsigned char));
 hipMemcpy(mappings, dmappings, 256*ntiles*sizeof(unsigned char),hipMemcpyDeviceToHost);
 hipMemcpyToSymbol(HIP_SYMBOL(const_mappings), mappings, ntiles*256*sizeof(unsigned char));

//STEP 2
  performAdaptiveEqualization<<<grid,block>>>(dimg_in, dimg_out, width, height);

hipDeviceSynchronize();
hipEventRecord(stop);

// WRITING OUTPUT IMAGE TO HOST MEMORY  
  hipMemcpy(img_out, dimg_out, height*width*sizeof(unsigned char),hipMemcpyDeviceToHost);

hipEventSynchronize(stop);
float milliseconds = 0;
hipEventElapsedTime(&milliseconds, start, stop);
std::cout  << "Time taken : " << milliseconds << std::endl;

// CLEANUP
  hipFree(dpdf);
  hipFree(dcdf);
  hipFree(dmappings);
  hipFree(dimg_out);
  hipFree(dimg_in);
  hipFree(const_mappings);
}
