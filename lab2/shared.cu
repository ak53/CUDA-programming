
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>


//#define LENGTH 100
//#define rowA 4
//#define colA 1
//#define rowB 1
//#define colB 4
#define w 100
#define tw 10
//#define TILE_BLOCKS 10
//#define TILE_WIDTH 100

using namespace std;

__global__ void mat_mult_simple(int (*a)[w], int (*b)[w], int (*c)[w]){
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        int result = 0;

        for (int i=0;i<w;i++){
                result += (a[row][i] * b[i][col]);
        }

        c[row][col] = result;
}

__global__ void mat_mult_shared(int (*a)[w], int (*b)[w], int (*c)[w]){
    __shared__ int s_a[tw][tw];
	__shared__ int s_b[tw][tw];
	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	int ty = threadIdx.y;
	int tx = threadIdx.x;

    int result = 0;
	
	for (int p=0;p<w/tw;p++){
		s_a[ty][tx] = a[row][p*tw+tx];
		s_b[ty][tx] = b[p*tw+ty][col];
		__syncthreads();

		for (int k=0;k<tw;k++){
			result += s_a[ty][k] * s_b[k][tx]; 
			__syncthreads();
		}	
	c[row][col] = result;
	}
}

int main(){

        int (*a)[w];
        int (*b)[w];
        int (*c)[w];

        int (*d_a)[w], (*d_b)[w], (*d_c)[w];
        //int *h_c;

        //int a_mat[rowA][colA] = {{0},{1},{2},{3}};
        //int b_mat[rowB][colB] = {0,1,2,3};
        //int c_mat[rowA][colB] = {};

        a = (int(*)[w])malloc(w * w *sizeof(int));
        b = (int(*)[w])malloc(w * w *sizeof(int));
        //h_c = (int*)malloc(rowA * colB * sizeof(int));



        for(int i=0 ; i< w; i++){
                for (int j=0;j<w;j++){
                        a[i][j] = 1;
                        b[i][j] = 1;
                }
        }


        hipMalloc((void**)&d_a, w*w*sizeof(int));
        hipMalloc((void**)&d_b, w*w*sizeof(int));
        hipMalloc((void**)&d_c, w*w*sizeof(int)); // host -> device

        c = (int(*)[w])malloc(w*w*sizeof(int)); //cpu device -> host

        hipMemcpy(d_a, a, w*w*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, w*w*sizeof(int), hipMemcpyHostToDevice);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        
        dim3 grid;
        grid.x = w/tw;
        grid.y = w/tw;
        dim3 block;
	block.x = tw;
        block.y = tw;
        hipEventRecord(start);
	//mat_mult_simple<<<grid,block>>>(d_a, d_b, d_c);
	mat_mult_shared<<<grid,block>>>(d_a, d_b, d_c);

        hipDeviceSynchronize();

        hipEventRecord(stop);


        hipMemcpy(c, d_c, w*w*sizeof(int), hipMemcpyDeviceToHost);

        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);


        for(int i=0; i<w ;i++){
        //        for (int j=0;j<w;j++){
       	                std::cout << c[99][i]<<"   ";
        //         }
                std::cout<<std::endl;
        }

        std::cout  << "Time taken : " << milliseconds << std::endl;
}


