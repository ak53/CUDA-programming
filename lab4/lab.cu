
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>

#define LENGTH 256
using namespace std;

struct SoA 
{ 
   int x[LENGTH];
   int y[LENGTH];
   int z[LENGTH]; 
};

struct S 
{ 
   int x;
   int y;
   int z; 
};

__global__ void add_soa(SoA* a, SoA* b, SoA* c){
    int i = threadIdx.x ;
	 if  (i < LENGTH){
	    c->x[i] = a->x[i] + b->x[i]; 
	    c->y[i] = a->y[i] + b->y[i]; 
	    c->z[i] = a->z[i] + b->z[i]; 
	}
}

__global__ void add_aos(S* a, S* b, S* c){
    int i = threadIdx.x ;
	 if  (i < LENGTH){
	    c[i].x = a[i].x + b[i].x; 
	    c[i].y = a[i].y + b[i].y; 
	    c[i].z = a[i].z + b[i].z; 
	}
}

int main(){

	SoA* a;
	SoA* b;
	SoA* c;

	S* d;
	S* e;
	S* f;

	a = (SoA*)malloc(sizeof(SoA));
	b = (SoA*)malloc(sizeof(SoA));
	c = (SoA*)malloc(sizeof(SoA));
	d = (S*)malloc(LENGTH*sizeof(S));
	e = (S*)malloc(LENGTH*sizeof(S));
	f = (S*)malloc(LENGTH*sizeof(S));

	for(int i=0 ; i< LENGTH; i++){
		a->x[i] = i;
		a->y[i] = i-1;
		a->z[i] = i-2;
		b->x[i] = i;
		b->y[i] = i-1;
		b->z[i] = i-2;
		d[i].x = i;
		d[i].y = i-1;
		d[i].z = i-2;
		e[i].x = i;
		e[i].y = i-1;
		e[i].z = i-2;
	}

	SoA* d_a;
	SoA* d_b;
	SoA* d_c;

	S* d_d;
	S* d_e;
	S* d_f;

	hipMalloc((void**)&d_a,sizeof(SoA));
	hipMalloc((void**)&d_b,sizeof(SoA));
	hipMalloc((void**)&d_c,sizeof(SoA));

	hipMalloc((void**)&d_d,LENGTH*sizeof(S));
	hipMalloc((void**)&d_e,LENGTH*sizeof(S));
	hipMalloc((void**)&d_f,LENGTH*sizeof(S));


	hipMemcpy(d_a,a,sizeof(SoA), hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,sizeof(SoA), hipMemcpyHostToDevice);

	hipMemcpy(d_d,d,LENGTH*sizeof(S), hipMemcpyHostToDevice);
	hipMemcpy(d_e,e,LENGTH*sizeof(S), hipMemcpyHostToDevice);

	// add_soa<<<1, LENGTH>>>(d_a,d_b,d_c); 
	add_aos<<<1, LENGTH>>>(d_d,d_e,d_f); 

	hipMemcpy(c,d_c,sizeof(SoA), hipMemcpyDeviceToHost);
	hipMemcpy(f,d_f,LENGTH*sizeof(S), hipMemcpyDeviceToHost);

	// for(int i=0 ; i< LENGTH; i++){
	// 	std::cout<<c->x[i]<<"  "<<c->y[i]<<"  "<<c->z[i]<<'\n'<<std::flush;
	// 	std::cout<<f[i].x<<"  "<<f[i].y<<"  "<<f[i].z<<'\n'<<std::flush;

	// }	

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(a);
	free(b);
	free(c);
	hipFree(d_d);
	hipFree(d_e);
	hipFree(d_f);
	free(d);
	free(e);
	free(f);
}

